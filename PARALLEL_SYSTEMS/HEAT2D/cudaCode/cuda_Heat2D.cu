#include "hip/hip_runtime.h"
#include <assert.h>
//#include <sys/time.h>
#include <time.h>

#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <cmath>
#include <algorithm>

using std::cout;
using std::endl;
using std::cerr;

#define DECLINE_HORIZONTAL 0.1
#define DECLINE_VERTICAL   0.1
#define STEPS              1000               /* number of time steps */

/* error check on hip macro */
#define CUDA_CHECK(command) \
{ \
  hipError_t status = command; \
  if(status != hipSuccess) \
  { \
    cerr << "Error : Cuda reports " << hipGetErrorString(status) << endl; \
    heatCleanExit(-6); \
  } \
}

/* Device(GPU) FUNCTIONS TO BE LAUNCHED AS KERNELS FROM HOST(CPU) */
/* This function is used to discover the local working area of each thread */
__global__ void heatDiscover(int * __restrict__ workRowS,
                             int * __restrict__ workRowE,
                             int * __restrict__ workColS,
                             int * __restrict__ workColE,
                             const int blockRows, const int blockCols,
                             const int threadsPerRow, const int threadsPerCol,
                             const int probCase,
                             const int gridRows, const int gridCols)
{
  const int thread_Id = blockIdx.x * blockDim.x + threadIdx.x;
  if(probCase == 1)//2 threads
  {
    if(thread_Id == 0)//west thread
    {
      workRowS[thread_Id] = 1;
      workRowE[thread_Id] = blockRows - 2;
      workColS[thread_Id] = 1;
      workColE[thread_Id] = blockCols - 1;
    }
    else//east thread
    {
      workRowS[thread_Id] = 1;
      workRowE[thread_Id] = blockRows - 2;
      workColS[thread_Id] = blockCols;
      workColE[thread_Id] = gridCols - 2;
    } 
  }
  else if(probCase == 2)//6, 8, 10, ... OR 4, 16, 64 ... threads
  {
    if(thread_Id == 0)//NW corner
    {
      workRowS[thread_Id] = 1;
      workRowE[thread_Id] = blockRows - 1;
      workColS[thread_Id] = 1;
      workColE[thread_Id] = blockCols - 1;
    }
    else if(thread_Id == (threadsPerCol - 1))//NE corner
    {
      workRowS[thread_Id] = 1;
      workRowE[thread_Id] = blockRows - 1;
      workColS[thread_Id] = gridCols - blockCols;
      workColE[thread_Id] = gridCols - 2;
    }
    else if(thread_Id == ((threadsPerRow * threadsPerCol) - threadsPerCol))//SW corner
    {
      workRowS[thread_Id] = gridRows - blockRows;
      workRowE[thread_Id] = gridRows - 2;
      workColS[thread_Id] = 1;
      workColE[thread_Id] = blockCols - 1;
    }
    else if(thread_Id == ((threadsPerCol * threadsPerRow) - 1))//SE corner
    {
      workRowS[thread_Id] = gridRows - blockRows;
      workRowE[thread_Id] = gridRows - 2;
      workColS[thread_Id] = gridCols - blockCols;
      workColE[thread_Id] = gridCols - 2;
    }
    else if(thread_Id < threadsPerCol)//NN side
    {
      workRowS[thread_Id] = 1;
      workRowE[thread_Id] = blockRows - 1;
      workColS[thread_Id] = thread_Id * blockCols;
      workColE[thread_Id] = workColS[thread_Id] + blockCols - 1;
    }
    else if((thread_Id > ((threadsPerCol * threadsPerRow) - threadsPerCol)) && 
            (thread_Id < ((threadsPerCol * threadsPerRow) - 1)))//SS side
    {
      workRowS[thread_Id] = gridRows - blockRows;
      workRowE[thread_Id] = gridRows - 2;
      workColS[thread_Id] = (thread_Id % threadsPerCol) * blockCols;
      workColE[thread_Id] =  workColS[thread_Id] + blockCols - 1;
    }
    else if((thread_Id % threadsPerCol) == 0)//WW side
    {
      workRowS[thread_Id] = (thread_Id / threadsPerCol) * blockRows;
      workRowE[thread_Id] = workRowS[thread_Id] + blockRows - 1;
      workColS[thread_Id] = 1;
      workColE[thread_Id] = blockCols - 1;
    }
    else if((thread_Id + 1) % threadsPerCol == 0)//EE side
    {
      workRowS[thread_Id] = ((thread_Id + 1 - threadsPerCol) / threadsPerCol) *
                              blockRows;
      workRowE[thread_Id] = workRowS[thread_Id] + blockRows - 1;
      workColS[thread_Id] = gridCols - blockCols;
      workColE[thread_Id] = gridCols - 2;
    }
    else//general case middle location
    {
      int rowMarginS = threadsPerCol;
      int rowMarginE = rowMarginS + threadsPerCol - 1;
      int rowOffset = 1;
      while(1)
      {
        if((thread_Id > rowMarginS) && (thread_Id < rowMarginE))
        {
          workRowS[thread_Id] = rowOffset * blockRows;
          break;
        }
        else
        {
          rowMarginS += threadsPerCol;
          rowMarginE += threadsPerCol;
          ++rowOffset;
        }
      }
      workRowE[thread_Id] = workRowS[thread_Id] + blockRows - 1;
      workColS[thread_Id] = (thread_Id % threadsPerCol) * blockCols;
      workColE[thread_Id] = workColS[thread_Id] + blockCols - 1;
    }
  }
}

/* This function updates the grid and is invoked on serial executions */
__global__ void heatUpdateSerial(const double * __restrict__ devOldHeatGrid,
                                       double * __restrict__ devNewHeatGrid,
                                 const int gridRows, const int gridCols)
{
  for(int i = 1; i < gridRows - 1; ++i)
  {
    for(int j = 1; j < gridCols - 1; ++j)
    {
      devNewHeatGrid[i*gridCols+j] = devOldHeatGrid[i*gridCols+j] +
                                    DECLINE_HORIZONTAL *
                                    (devOldHeatGrid[(i+1)*gridCols+j] +
                                    devOldHeatGrid[(i-1)*gridCols+j] -
                                    (2 * devOldHeatGrid[i*gridCols+j])) +
                                    DECLINE_VERTICAL *
                                    (devOldHeatGrid[i*gridCols+j+1] +
                                    devOldHeatGrid[i*gridCols+j-1] -
                                    (2 * devOldHeatGrid[i*gridCols+j]));
    }
  }    
}

/* This function updates the grid and is invoked on parallel executions */
__global__ void heatUpdateParallel(const double * __restrict__ devOldHeatGrid,
                                   double       * __restrict__ devNewHeatGrid,
                                   const int    * __restrict__ workRowS,
                                   const int    * __restrict__ workRowE,
                                   const int    * __restrict__ workColS,
                                   const int    * __restrict__ workColE,
                                   const int                   gridCols)
{
  const int thread_Id = blockIdx.x * blockDim.x + threadIdx.x;
  //get the borders in registers for 1 cycle memory access
  const int wRowS = workRowS[thread_Id];
  const int wRowE = workRowE[thread_Id];
  const int wColS = workColS[thread_Id];
  const int wColE = workColE[thread_Id];

  for(int i = wRowS; i <= wRowE; ++i)
  {
    for(int j = wColS; j <= wColE; ++j)
    {
      devNewHeatGrid[i*gridCols+j] = devOldHeatGrid[i*gridCols+j] +
                                 DECLINE_HORIZONTAL *
                                 (devOldHeatGrid[(i+1)*gridCols+j] +
                                 devOldHeatGrid[(i-1)*gridCols+j] -
                                 (2*devOldHeatGrid[i*gridCols+j])) +
                                 DECLINE_VERTICAL *
                                 (devOldHeatGrid[i*gridCols+j+1] +
                                 devOldHeatGrid[i*gridCols+j-1] -
                                 (2*devOldHeatGrid[i*gridCols+j]));
    }
  }
}                                      

//declare dynamic variables
double * heatGrid = nullptr;
double * devNewHeatGrid = nullptr;
double * devOldHeatGrid = nullptr;
int * workRowS = nullptr;
int * workRowE = nullptr;
int * workColS = nullptr;
int * workColE = nullptr;
int * devWorkRowS = nullptr;
int * devWorkRowE = nullptr;
int * devWorkColS = nullptr;
int * devWorkColE = nullptr;

/* HOST(CPU) FUNCTIONS */
/* This function intialises the temperature on the given grid with higher
   temperatures at the centre, progressively lower ones until the sides 
   and 0s at the perimetre 
*/
static inline void heatInit(double * heatGrid,
                            const int gridRows, const int gridCols)
{
  for(int i = 0; i < gridRows; ++i)//avoid halo area
    for(int j = 0; j < gridCols; ++j)
      heatGrid[i*gridCols+j] = (double) (i * (gridRows - i - 1) * j * (gridCols - j - 1));
}

/* This function writes out the input grid to a .dat file in current path */
static inline int heatWrite(const double * heatGrid, const int flag,
                            const int gridRows, const int gridCols,
                            const int threadsPerBlock, const int blocksPerGrid)
{
  char filePath[70] = "";
  if(flag == 0)
  {
    sprintf(filePath, "%d_%d_cuda_%d_%d_Initial.dat", gridRows, gridCols,
            threadsPerBlock, blocksPerGrid);
  }
  else
  {
    sprintf(filePath, "%d_%d_cuda_%d_%d_Final.dat", gridRows, gridCols,
            threadsPerBlock, blocksPerGrid);
  }

  FILE * fp = fopen(filePath, "w");
  if(fp == nullptr)
    return -1;
  for(int i = 0; i < gridRows; ++i)
  {
    for(int j = 0; j < gridCols; ++j)
    {
      fprintf(fp, "%-.1lf", fabs(heatGrid[i*gridCols+j]));//some 0.0s appear as -0.0s
      if(j != (gridCols - 1))
        fprintf(fp, " ");
    }
    fprintf(fp, "\n");
  }
  fclose(fp);
  return 0;//all ok
}

/* This function swaps between the 2 grids to avoid assignmenets */
static inline void heatSwap(double ** a, double ** b)
{
  double *temp = *a;
  *a = *b;
  *b = temp;
}

/* This function cleans up memory to prevent leaks on any exit error */
static inline void heatCleanExit(const int errorCode)
{
  if(heatGrid != nullptr)
  {
    free(heatGrid);
    heatGrid = nullptr;
  }
  if(devOldHeatGrid != nullptr)
  {
    CUDA_CHECK(hipFree(devOldHeatGrid));
    devOldHeatGrid = nullptr;
  }
  if(devNewHeatGrid != nullptr)
  {
    CUDA_CHECK(hipFree(devNewHeatGrid));
    devNewHeatGrid = nullptr;
  }
  if(workRowS != nullptr)
  {
    free(workRowS);
    workRowS = nullptr;
  }
  if(devWorkRowS != nullptr)
  {
    CUDA_CHECK(hipFree(devWorkRowS));
    devWorkRowS = nullptr;
  }
  if(workRowE != nullptr)
  {
    free(workRowE);
    workRowE = nullptr;
  }
  if(devWorkRowE != nullptr)
  {
    CUDA_CHECK(hipFree(devWorkRowE));
    devWorkRowE = nullptr;
  }
  if(workColS != nullptr)
  {
    free(workColS);
    workColS = nullptr;
  }
  if(devWorkColS != nullptr)
  {
    CUDA_CHECK(hipFree(devWorkColS));
    devWorkColS = nullptr;
  }
  if(workColE != nullptr)
  {
    free(workColE);
    workColE = nullptr;
  }
  if(devWorkColE != nullptr)
  {
    CUDA_CHECK(hipFree(devWorkColE));
    devWorkColE = nullptr;
  }
  exit(errorCode);
}

/* Main program function */
int main(int argc, char *argv[])
{
  //get the properties
  hipDeviceProp_t deviceProp;
  CUDA_CHECK(hipGetDeviceProperties(&deviceProp, 0));
  cout << "GPU PROPERTIES\n";
  cout << "******************************************************************\n";
  cout << "Cuda Device prop succeeded" << endl;
  cout << "System minor " << deviceProp.minor << endl;
  cout << "System major " << deviceProp.major << endl;
  cout << "Agent Prop Name " << deviceProp.name << endl;
  cout << "Total Global Memory " << deviceProp.totalGlobalMem << " bytes\n";
  cout << "Shared Memory Per Block " << deviceProp.sharedMemPerBlock << " bytes\n";
  cout << "Registers per block " << deviceProp.regsPerBlock << endl;
  cout << "Warp size " << deviceProp.warpSize << endl;
  cout << "Max Threads Per Block " << deviceProp.maxThreadsPerBlock << endl;
  cout << "Max clock frequency of the multiProcessors " << deviceProp.clockRate << " kHz\n";
  cout << "Size of shared memory region " << deviceProp.totalConstMem << " bytes\n";
  cout << "Number of multi-processors (compute units) " << deviceProp.multiProcessorCount << endl;
  cout << "******************************************************************\n\n";
  //get properties to check on input data possible run scenarios
  const int devMaxThreadsPerBlock = deviceProp.maxThreadsPerBlock;
  const int devMaxConcurrentThreads = deviceProp.multiProcessorCount *
                                      deviceProp.warpSize;

  //get the command line input data and do initial checks
  if(argc != 5)
  {
    cerr << "Not enough input data, need 4\n";
    cerr << "Grid_Rows Grid_Collumns Threads_Per_Block Blocks_Per_Grid\n";
    cerr << "Aborting...\n";
    heatCleanExit(-1);
  }
  const int gridRows = atoi(argv[1]);
  const int gridCols = atoi(argv[2]);
  const int threadsPerBlock = atoi(argv[3]);
  const int blocksPerGrid = atoi(argv[4]);
  const int gridSize = gridRows * gridCols;
  const int totalThreads = threadsPerBlock * blocksPerGrid;
  if(gridRows < 0 || gridCols < 0 || threadsPerBlock < 1 || blocksPerGrid < 1)
  {
    cerr << "Invalid Input Data\n";
    cerr << "Grid Rows = " << gridRows << endl;
    cerr << "Grid Cols = " << gridCols << endl;
    cerr << "Threads Per Block = " << threadsPerBlock << endl;
    cerr << "Blocks Per Grid = " << blocksPerGrid << endl;
    cerr << "Aborting...\n";
    heatCleanExit(-1);
  }
  //do checks based on device(GPU) capabilities
  if(threadsPerBlock > devMaxThreadsPerBlock)
  {
    cerr << "Maximum threads per block exceeded for current device\n";
    cerr << "Aborting...\n";
    heatCleanExit(-2);
  }
  if(totalThreads > devMaxConcurrentThreads)
  {
    cerr << "Maximum concurrent threads exceeded for current device\n";
    cerr << "Aborting...\n";
    heatCleanExit(-2);
  }

  //allocate host(CPU) memory
  heatGrid = (double *) malloc(gridSize * sizeof(double));
  if(heatGrid == nullptr)
  {
    cerr << "Error, not enough memory...\nAborting...\n";
    heatCleanExit(1);
  }

  //initialise with 0.0s the heat grid
  for(int i = 0; i < gridRows; ++i)
    for(int j = 0; j < gridCols; ++j)
      heatGrid[i*gridCols+j] = 0.0;

  //allocate device(GPU) global memory
  CUDA_CHECK(hipMalloc((void **)&devNewHeatGrid, gridSize * sizeof(double)));

  //transfer data from host(CPU) to device(GPU) memory
  CUDA_CHECK(hipMemcpy(devNewHeatGrid, heatGrid, gridSize * sizeof(double),
                      hipMemcpyHostToDevice));

  //initialise the heat grid with actual data
  heatInit(heatGrid, gridRows, gridCols);

  //allocate device(GPU) global memory
  CUDA_CHECK(hipMalloc((void **)&devOldHeatGrid, gridSize * sizeof(double)));

  //transfer data from host(CPU) to device(GPU) memory
  CUDA_CHECK(hipMemcpy(devOldHeatGrid, heatGrid, gridSize * sizeof(double),
                      hipMemcpyHostToDevice));

  //write out the initial grid to the corresponding file
  // if(heatWrite(heatGrid, 0, gridRows, gridCols,
               // threadsPerBlock, blocksPerGrid) == -1)
  // {
    // cerr << "Error, could not create the initial file...\nAborting...\n";
    // heatCleanExit(2);
  // }

  //calculate the kernel dimensions (x,y,z) threads/block and blocks/grid
  dim3 cudaThreads(threadsPerBlock, 1, 1);
  dim3 cudaBlocks(blocksPerGrid, 1 ,1);

  //define the timer structs to be used
  // struct timespec start;
  // struct timespec end;
  // double totalTime = 0.0;

  if(totalThreads == 1)//serial execution
  {
    cout << "Serial execution with 1 cuda thread\n";

    //start the timer
    //clock_gettime(CLOCK_MONOTONIC, &start);
	clock_t begin = clock();

    //solve the problem
    for(int steps = 0; steps < STEPS; ++steps)
    {
      //launch the kernel
      heatUpdateSerial<<<cudaBlocks, cudaThreads>>>(devOldHeatGrid, devNewHeatGrid,
                                                    gridRows, gridCols);

      //wait for device(GPU) to finish it's work
      CUDA_CHECK(hipDeviceSynchronize());

      //old = new
      heatSwap(&devNewHeatGrid, &devOldHeatGrid);
    }

    //stop the timer and print the result
    //clock_gettime(CLOCK_MONOTONIC, &end);
	clock_t end = clock();
	double totalTime = (double)(end - begin) / CLOCKS_PER_SEC;
    //totalTime = ((end.tv_sec - start.tv_sec) * 1000.0) +
    //            ((end.tv_nsec - start.tv_nsec) / 1000000.0); 
    cout << "\nElapsed time was " << totalTime << " ms\n";
  }
  else//parallel execution
  {
    //initial check on number of threads
    if(totalThreads % 2 != 0)
    {
      cout << "Can't parition grid fairly with odd number of threads = "
           << totalThreads
           << "\nAborting...\n";
      heatCleanExit(4);
    }
    cout << "Parallel execution with ";
    cout << "Threads Per Block : " << threadsPerBlock << endl;
    cout << "Blocks Per Grid : " << blocksPerGrid << endl;
    cout << "Total Cuda Threads : " << totalThreads << endl;

    //allocate arrays for neighbour discovery 
    //working border rows
    workRowS = (int *) malloc(totalThreads * sizeof(int));
    if(workRowS == nullptr)
    {
      cerr << "Error, not enough memory...\nAborting...\n";
      heatCleanExit(5);
    }
    workRowE = (int *) malloc(totalThreads * sizeof(int));
    if(workRowE == nullptr)
    {
      cerr << "Error, not enough memory...\nAborting...\n";
      heatCleanExit(5);
    }
    //working border collumns
    workColS = (int *) malloc(totalThreads * sizeof(int));
    if(workColS == nullptr)
    {
      cerr << "Error, not enough memory...\nAborting...\n";
      heatCleanExit(5);
    }
    workColE = (int *) malloc(totalThreads * sizeof(int));
    if(workColE == nullptr)
    {
      cerr << "Error, not enough memory...\nAborting...\n";
      heatCleanExit(5);
    }
    
    int blockRows = 0;//total rows for each block of threads data block
    int blockCols = 0;//total cols for each block of threads data block
    int threadsPerRow = 0, threadsPerCol = 0;//vertical and horizontal distrib

    //classify problem cases based on total threads
    int cut = (int) sqrt(totalThreads);
    double cutF = sqrt(totalThreads);
    int probCase;
    if(totalThreads == 2)//case 1 : handling 2 threads
    {
      blockRows = gridRows;
      blockCols = gridCols / totalThreads;
      threadsPerRow = gridRows / blockRows;
      threadsPerCol = gridCols / blockCols;
      if((threadsPerRow * threadsPerCol) != totalThreads)
      {
        cout << "Grid partitioning results to remains...\nAborting...\n";
        heatCleanExit(6);
      }
      probCase = 1;
    }
    else if(cutF > (double) cut)//case 2.1 : handling 6, 8, 10, ... threads
    {
      if(gridSize % totalThreads != 0)//can't cut it without remains
      {
        cout << "Grid partitioning results to remains...\nAborting...\n";
        heatCleanExit(7);
      }
      const int localProbSize = gridSize / totalThreads;
      int spread = gridSize;
      //find the best possible partition
      for(int i = gridRows; i > 0; --i)//priority to rows
      {
        for(int j = gridCols; j > 0; --j)
        {
          if((i * j) == localProbSize)
          {
            if(gridRows % i != 0 || gridCols % j != 0)
              continue;
            if(abs(i -j) < spread)
            {
              spread = abs(i - j);
              blockRows = i;
              blockCols = j;
            }
          }
        }
      }
      threadsPerRow = gridRows / blockRows;
      threadsPerCol = gridCols / blockCols;
      if((threadsPerRow * threadsPerCol) != totalThreads)
      {
        cout << "Grid partitioning results to remains...\nAborting...\n";
        heatCleanExit(7);
      }
      probCase = 2;
    }
    else//case 2.2 : handling 4, 9, 16, ... threads
    {
      threadsPerRow = cut;
      threadsPerCol = cut;
      if((gridRows % cut != 0) || (gridCols % cut != 0))//can't cut even blocks
      {
        cout << "Grid partitioning results to remains...\nAborting...\n";
        heatCleanExit(8);
      }
      blockRows = gridRows / cut;
      blockCols = gridCols / cut;
      probCase = 2;
    }

    printf("Grid can be partioned without remains...\n"
             "Rows per block : %d, Columns per block : %d\n"
             "Vertical threads : %d, Horizontal threads : %d\n\n",
             blockRows, blockCols, threadsPerRow, threadsPerCol);

    //working and global discovery phase
    //allocate device(GPU) global memory
    CUDA_CHECK(hipMalloc((void **)&devWorkRowS, totalThreads * sizeof(int)));
    CUDA_CHECK(hipMalloc((void **)&devWorkRowE, totalThreads * sizeof(int)));
    CUDA_CHECK(hipMalloc((void **)&devWorkColS, totalThreads * sizeof(int)));
    CUDA_CHECK(hipMalloc((void **)&devWorkColE, totalThreads * sizeof(int)));
    //transfer data from host(CPU) to device(GPU) memory
    CUDA_CHECK(hipMemcpy(devWorkRowS, workRowS, totalThreads * sizeof(int),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(devWorkRowE, workRowE, totalThreads * sizeof(int),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(devWorkColS, workColS, totalThreads * sizeof(int),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(devWorkColE, workColE, totalThreads * sizeof(int),
                          hipMemcpyHostToDevice));

    heatDiscover<<<cudaBlocks, cudaThreads>>>
                (devWorkRowS, devWorkRowE, devWorkColS, devWorkColE,
                 blockRows, blockCols,
                 threadsPerRow, threadsPerCol, probCase,
                 gridRows, gridCols);

    //wait for device(GPU) to finish it's work
    CUDA_CHECK(hipDeviceSynchronize());

    //start the timer
    // clock_gettime(CLOCK_MONOTONIC, &start);
	clock_t begin = clock();

    //launch the kernel
    for(int steps = 0; steps < STEPS; ++steps)
    {
      heatUpdateParallel<<<cudaBlocks, cudaThreads>>>
                        (devOldHeatGrid, devNewHeatGrid,
                         devWorkRowS, devWorkRowE, devWorkColS, devWorkColE,
                         gridCols);

      //wait for device(GPU) to finish it's work
      CUDA_CHECK(hipDeviceSynchronize());

      //old = new
      heatSwap(&devNewHeatGrid, &devOldHeatGrid);
    }
    
    //stop the timer and print the result
	clock_t end = clock();
	double totalTime = (double)(end - begin) / CLOCKS_PER_SEC;
    // clock_gettime(CLOCK_MONOTONIC, &end);
    // totalTime = ((end.tv_sec - start.tv_sec) * 1000.0) +
                // ((end.tv_nsec - start.tv_nsec) / 1000000.0); 
    cout << "\nElapsed time was " << totalTime << " ms\n";
  }

  if(STEPS % 2 == 0)//get the correct version
  {
    CUDA_CHECK(hipMemcpy(heatGrid, devOldHeatGrid, gridSize * sizeof(double),
                        hipMemcpyDeviceToHost));
  }
  else
  {
    CUDA_CHECK(hipMemcpy(heatGrid, devNewHeatGrid, gridSize * sizeof(double),
                        hipMemcpyDeviceToHost));
  }
  
  //write out the final grid to the corresponding file
  // if(heatWrite(heatGrid, 1, gridRows, gridCols,
               // threadsPerBlock, blocksPerGrid) == -1)
  // {
    // cerr << "Error, could not create the initial file...\nAborting...\n";
    // heatCleanExit(3);
  // }

  //clear memory and exit
  heatCleanExit(0);
}